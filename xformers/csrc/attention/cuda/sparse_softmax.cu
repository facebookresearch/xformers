#include "hip/hip_runtime.h"
// This file was modified from sputnik to implement batch support for
// sparse softmax directly in the kernels
//
// Copyright 2020 The Sputnik Authors.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cmath>

#include "sputnik/cuda_utils.h"
#include "sputnik/load_store.h"

#include <ATen/ATen.h>
#include <torch/types.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

namespace sputnik {

namespace {

__global__ void SparseSoftmaxKernel(
    int m,
    int n,
    const float* __restrict__ values,
    const int* __restrict__ row_indices,
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    float* __restrict__ output_values,
    int nnz) {
  // Calculate the index of the row that this block will process.
  int m_index = blockIdx.x * blockDim.y + threadIdx.y;
  if (m_index >= m)
    return;
  m_index = Load(row_indices + m_index);

  // Load the row offset and calculate the number of non-zeros in
  // the row.
  int row_offset = Load(row_offsets + m_index);
  int nonzeros = Load(row_offsets + m_index + 1) - row_offset;

  int batch_offset = blockIdx.y * nnz;

  // Step 1: Find the maximum value in our row.
  const float* in = values + row_offset + batch_offset;
  float max = -INFINITY;
  for (int idx = threadIdx.x; idx < nonzeros; idx += blockDim.x) {
    float x = Load(in + idx);
    max = x > max ? x : max;
  }
  for (int idx = 1; idx < blockDim.x; idx *= 2) {
    float x = __shfl_xor_sync(0xffffffff, max, idx);
    max = x > max ? x : max;
  }

  // Step 2: Compute the normalization constant. Invert the norm
  // once so we don't need to do repeated division.
  float norm = 0.0f;
  for (int idx = threadIdx.x; idx < nonzeros; idx += blockDim.x) {
    norm += expf(Load(in + idx) - max);
  }
  for (int idx = 1; idx < blockDim.x; idx *= 2) {
    norm += __shfl_xor_sync(0xffffffff, norm, idx);
  }
  norm = 1.0f / norm;

  // step 3: Normalize the exponentials of the input and store the
  // results.
  float* out = output_values + row_offset + batch_offset;
  for (int idx = threadIdx.x; idx < nonzeros; idx += blockDim.x) {
    Store(expf(Load(in + idx) - max) * norm, out + idx);
  }
}

} // namespace

hipError_t SparseSoftmax(
    int m,
    int n,
    int nonzeros,
    const float* __restrict__ values,
    const int* __restrict__ row_indices,
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    float* __restrict__ output_values,
    hipStream_t stream,
    int batch) {
  // NOTE: SparseSoftmaxKernel currently only supports 1 warp per row
  // of the input matrix. We launch two warps per block, with each
  // mapped to different rows to enable us to hit max occupancy.
  constexpr int kBlockWidth = 32;
  constexpr int kWarpsPerBlock = 2;
  dim3 grid_dim(std::ceil(static_cast<float>(m) / kWarpsPerBlock), batch);
  dim3 block_dim(kBlockWidth, kWarpsPerBlock);

  SparseSoftmaxKernel<<<grid_dim, block_dim, 0, stream>>>(
      m,
      n,
      values,
      row_indices,
      row_offsets,
      column_indices,
      output_values,
      nonzeros);
  return hipGetLastError();
}

} // namespace sputnik

at::Tensor sparse_softmax_sputnik(
    int64_t m,
    int64_t n,
    const at::Tensor& row_indices,
    const at::Tensor& values,
    const at::Tensor& row_offsets,
    const at::Tensor& column_indices) {
  TORCH_CHECK(values.dim() == 2);
  TORCH_CHECK(row_indices.dim() == 1);
  TORCH_CHECK(row_offsets.dim() == 1);
  TORCH_CHECK(column_indices.dim() == 1);
  TORCH_CHECK(values.size(1) == column_indices.size(0));

  TORCH_CHECK(row_indices.is_cuda(), "row_indices must be a CUDA tensor");
  TORCH_CHECK(values.is_cuda(), "values must be a CUDA tensor");
  TORCH_CHECK(row_offsets.is_cuda(), "row_offsets must be a CUDA tensor");
  TORCH_CHECK(column_indices.is_cuda(), "column_offsets must be a CUDA tensor");

  TORCH_CHECK(
      row_indices.is_contiguous(), "row_indices must be a contiguous tensor");
  TORCH_CHECK(values.is_contiguous(), "values must be a contiguous tensor");
  TORCH_CHECK(
      row_offsets.is_contiguous(), "row_offsets must be a contiguous tensor");
  TORCH_CHECK(
      column_indices.is_contiguous(),
      "column_offsets must be a contiguous tensor");

  TORCH_CHECK(!row_indices.is_sparse(), "row_indices must be a dense tensor");
  TORCH_CHECK(!values.is_sparse(), "values must be a dense tensor");
  TORCH_CHECK(!row_offsets.is_sparse(), "row_offsets must be a dense tensor");
  TORCH_CHECK(
      !column_indices.is_sparse(), "column_offsets must be a dense tensor");

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int batch = values.size(0);
  int nonzeros = column_indices.size(0);

  at::Tensor output = at::empty({batch, nonzeros}, values.options());

  AT_CUDA_CHECK(
      sputnik::SparseSoftmax(
          m,
          n,
          nonzeros,
          values.data_ptr<float>(),
          row_indices.data_ptr<int>(),
          row_offsets.data_ptr<int>(),
          column_indices.data_ptr<int>(),
          output.data_ptr<float>(),
          stream,
          batch));

  return output;
}

// Taken from sputnik SparseSoftmax with minor modifications
// to adapt it to perform the backward operation
__global__ void SparseSoftmaxBackwardKernel(
    int m,
    int n,
    const float* __restrict__ gradient,
    const float* __restrict__ values,
    const int* __restrict__ row_indices,
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    float* __restrict__ output_values,
    int nnz) {
  // Calculate the index of the row that this block will process.
  int m_index = blockIdx.x * blockDim.y + threadIdx.y;
  if (m_index >= m)
    return;
  m_index = sputnik::Load(row_indices + m_index);

  // Load the row offset and calculate the number of non-zeros in
  // the row.
  int row_offset = sputnik::Load(row_offsets + m_index);
  int nonzeros = sputnik::Load(row_offsets + m_index + 1) - row_offset;

  int batch_offset = blockIdx.y * nnz;

  const float* in = values + row_offset + batch_offset;
  const float* grad = gradient + row_offset + batch_offset;

  // Step 1: Compute the intermediate sum used for the gradient
  float sum = 0.0f;
  for (int idx = threadIdx.x; idx < nonzeros; idx += blockDim.x) {
    sum += sputnik::Load(in + idx) * sputnik::Load(grad + idx);
  }
  for (int idx = 1; idx < blockDim.x; idx *= 2) {
    sum += __shfl_xor_sync(0xffffffff, sum, idx);
  }

  // step 2: Compute the gradients
  float* out = output_values + row_offset + batch_offset;
  for (int idx = threadIdx.x; idx < nonzeros; idx += blockDim.x) {
    sputnik::Store(
        sputnik::Load(in + idx) * (sputnik::Load(grad + idx) - sum), out + idx);
  }
}

at::Tensor sparse_softmax_backward_sputnik(
    int64_t m,
    int64_t n,
    const at::Tensor& row_indices,
    const at::Tensor& values,
    const at::Tensor& grad,
    const at::Tensor& row_offsets,
    const at::Tensor& column_indices) {
  TORCH_CHECK(grad.dim() == 2);
  TORCH_CHECK(values.dim() == 2);
  TORCH_CHECK(row_indices.dim() == 1);
  TORCH_CHECK(row_offsets.dim() == 1);
  TORCH_CHECK(column_indices.dim() == 1);
  TORCH_CHECK(values.size(1) == column_indices.size(0));
  TORCH_CHECK(values.size(0) == grad.size(0));
  TORCH_CHECK(values.size(1) == grad.size(1));

  TORCH_CHECK(grad.is_cuda(), "grad must be a CUDA tensor");
  TORCH_CHECK(row_indices.is_cuda(), "row_indices must be a CUDA tensor");
  TORCH_CHECK(values.is_cuda(), "values must be a CUDA tensor");
  TORCH_CHECK(row_offsets.is_cuda(), "row_offsets must be a CUDA tensor");
  TORCH_CHECK(column_indices.is_cuda(), "column_offsets must be a CUDA tensor");

  TORCH_CHECK(grad.is_contiguous(), "grad must be a contiguous tensor");
  TORCH_CHECK(
      row_indices.is_contiguous(), "row_indices must be a contiguous tensor");
  TORCH_CHECK(values.is_contiguous(), "values must be a contiguous tensor");
  TORCH_CHECK(
      row_offsets.is_contiguous(), "row_offsets must be a contiguous tensor");
  TORCH_CHECK(
      column_indices.is_contiguous(),
      "column_offsets must be a contiguous tensor");

  TORCH_CHECK(!grad.is_sparse(), "grad must be a dense tensor");
  TORCH_CHECK(!row_indices.is_sparse(), "row_indices must be a dense tensor");
  TORCH_CHECK(!values.is_sparse(), "values must be a dense tensor");
  TORCH_CHECK(!row_offsets.is_sparse(), "row_offsets must be a dense tensor");
  TORCH_CHECK(
      !column_indices.is_sparse(), "column_offsets must be a dense tensor");

  TORCH_CHECK(
      values.device() == grad.device(),
      "values should be in the same device as grad");
  TORCH_CHECK(
      values.device() == row_indices.device(),
      "a should be in the same device as row_indices");
  TORCH_CHECK(
      values.device() == row_offsets.device(),
      "a should be in the same device as row_offsets");
  TORCH_CHECK(
      values.device() == column_indices.device(),
      "a should be in the same device as column_indices");

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int batch = values.size(0);
  int nonzeros = column_indices.size(0);

  at::Tensor output = at::empty({batch, nonzeros}, values.options());

  // NOTE: SparseSoftmaxBackwardKernel currently only supports 1 warp per row
  // of the input matrix. We launch two warps per block, with each
  // mapped to different rows to enable us to hit max occupancy.
  constexpr int kBlockWidth = 32;
  constexpr int kWarpsPerBlock = 2;
  dim3 grid_dim(std::ceil(static_cast<float>(m) / kWarpsPerBlock), batch);
  dim3 block_dim(kBlockWidth, kWarpsPerBlock);

  SparseSoftmaxBackwardKernel<<<grid_dim, block_dim, 0, stream>>>(
      m,
      n,
      grad.data_ptr<float>(),
      values.data_ptr<float>(),
      row_indices.data_ptr<int>(),
      row_offsets.data_ptr<int>(),
      column_indices.data_ptr<int>(),
      output.data_ptr<float>(),
      nonzeros);
  AT_CUDA_CHECK(hipGetLastError());

  return output;
}

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::sparse_softmax_sputnik"),
      TORCH_FN(sparse_softmax_sputnik));
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::sparse_softmax_backward_sputnik"),
      TORCH_FN(sparse_softmax_backward_sputnik));
}
