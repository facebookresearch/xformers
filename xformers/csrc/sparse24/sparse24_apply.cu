#include "hip/hip_runtime.h"
#include <ATen/ScalarOps.h>
#include <ATen/Tensor.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/library.h>
#include "sparse24_metadata.h"
#include "sparse24_pack.h"

using namespace xformers::sp24;

namespace {

template <typename KT>
__global__ void __launch_bounds__(32 /* num_threads */)
    sparse24_apply_kernel(typename KT::Params p) {
  KT::sparse24_apply_kernel(p);
}

// Apply a 2:4 sparsify pattern computed with
// `sparse24_sparsify_both_ways_kernel` to another Tensor
template <typename Element, typename MetadataFormat, bool kIsMeta>
std::
    tuple<
        at::Tensor, // packed
        at::Tensor, // packed_meta_reordered
        at::Tensor, // packed_trans
        at::Tensor // packed_trans_meta_reordered
        >
    sparse24_apply_typed(
        at::Tensor input, // Tensor to sparsify
        at::Tensor threads_masks // Returned by `sparse24_sparsify_both_ways`
    ) {
  using KT = KernelTypes<Element>;
  // TODO: Technically we should be able to deal with that
  // by running on the transpose of `input` and swapping
  // `packed` & `packed_t`.
  // This would require to adapt the `threads_masks` a bit tho.
  if (input.stride(1) != 1) {
    input = input.contiguous();
  }
  std::optional<at::cuda::CUDAGuard> device_guard;
  if (!kIsMeta) {
    device_guard.emplace(input.device());
  }

  TORCH_CHECK(input.dim() == 2);
  TORCH_CHECK(input.stride(1) == 1);
  TORCH_CHECK(input.stride(0) % 8 == 0);
  TORCH_CHECK(input.size(1) % 32 == 0, "Wrong alignment shape[1]");

  auto rows = input.size(0);
  auto cols = input.size(1);

  auto [compressed, packed, packed_meta_reordered] =
      MetadataFormat::create_compressed_representation(
          rows, cols, input, false);
  auto [compressed_trans, packed_trans, packed_trans_meta_reordered] =
      MetadataFormat::create_compressed_representation(
          cols, rows, input, false);

  typename KT::Params p;
  p.input_s0 = input.stride(0);
  p.input_dim0 = input.size(0);
  p.input_dim1 = input.size(1);

  p.packed_stride = packed.stride(0);
  p.packed_trans_stride = packed_trans.stride(0);

  if (!kIsMeta) {
    p.input = (Element const*)input.data_ptr();
    p.packed = (Element*)packed.data_ptr();
    p.packed_trans = (Element*)packed_trans.data_ptr();
    p.threads_masks = (uint64_t*)threads_masks.data_ptr();
  }

  TORCH_CHECK(threads_masks.dim() == 3);
  TORCH_CHECK(
      threads_masks.size(0) == p.getBlocksGrid().x * p.getThreadsGrid().x);
  TORCH_CHECK(
      threads_masks.size(1) == p.getBlocksGrid().y * p.getThreadsGrid().y);
  TORCH_CHECK(threads_masks.stride(1) == sizeof(p.threads_masks[0]));
  TORCH_CHECK(threads_masks.size(2) == sizeof(p.threads_masks[0]));
  TORCH_CHECK(threads_masks.stride(2) == 1);
  TORCH_CHECK(threads_masks.scalar_type() == at::ScalarType::Byte);

  if (!kIsMeta) {
    size_t smem_bytes = 0;
    sparse24_apply_kernel<KT>
        <<<p.getBlocksGrid(),
           p.getThreadsGrid(),
           smem_bytes,
           at::cuda::getCurrentCUDAStream()>>>(p);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
  return std::make_tuple(
      compressed,
      packed_meta_reordered,
      compressed_trans,
      packed_trans_meta_reordered);
}

template <bool kIsMeta>
std::
    tuple<
        at::Tensor, // packed
        at::Tensor, // packed_meta_reordered
        at::Tensor, // packed_trans
        at::Tensor // packed_trans_meta_reordered
        >
    sparse24_apply(
        at::Tensor input, // Tensor to sparsify
        at::Tensor threads_masks, // Returned by `sparse24_sparsify_both_ways`
        std::string backend) {
  auto runTyped = [&](auto type) {
    using ElementT = decltype(type);
    if (backend == "cusparselt") {
      return sparse24_apply_typed<ElementT, MetadataCuSparseLtSm80, kIsMeta>(
          input, threads_masks);
    } else {
      TORCH_CHECK(
          backend == "cutlass",
          "backend argument only supports `cutlass` or `cusparselt`");
      return sparse24_apply_typed<ElementT, MetadataCutlassSm80, kIsMeta>(
          input, threads_masks);
    }
  };

  if (input.scalar_type() == at::ScalarType::Half) {
    return runTyped(cutlass::half_t());
  } else {
    TORCH_CHECK(
        input.scalar_type() == at::ScalarType::Half ||
        input.scalar_type() == at::ScalarType::BFloat16);
    return runTyped(cutlass::bfloat16_t());
  }
}

} // namespace

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::sparse24_apply"),
      TORCH_FN(sparse24_apply<false>));
}

TORCH_LIBRARY_IMPL(xformers, Meta, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::sparse24_apply"),
      TORCH_FN(sparse24_apply<true>));
}
